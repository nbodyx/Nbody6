#include "hip/hip_runtime.h"
#include <cstdio>
#include <cmath>
#include <cassert>
#include <cstdlib>
#include <omp.h>
#ifdef WITH_CUDA5
#  include <hip/hip_runtime_api.h>
#  define CUDA_SAFE_CALL checkCudaErrors
#else
#  include <cutil.h>
#endif
#include "cuda_pointer.h"

#define NTHREAD 64 // 64 or 128
// #define NJBLOCK 14 // for GTX 470
// #define NJBLOCK 28 // for GTX660Ti 

#if 0 // V100?
#  define NJBLOCK   80
#  define NXREDUCE 128
#elif 1 // P100?
#  define NJBLOCK  56
#  define NXREDUCE 64
#else // safe version
#  define NJBLOCK  28
#  define NXREDUCE 32
#endif

#define NIBLOCK 32 // 16 or 32 
#define NIMAX (NTHREAD * NIBLOCK) // 2048

// #define NXREDUCE 32 // must be 2^n such that >NJBLOCK
#define NYREDUCE  (256/NXREDUCE)

#define NNB_PER_BLOCK 256 // NNB per block, must be power of 2
#define NB_BUF_SIZE (1<<20)
// #define NNB_MAX       384 // total NNB at reduced

#define MAX_CPU 16
#define MAX_GPU 4

// for clearity, for myself
#define __out

#define PROFILE

#define NAN_CHECK(val) assert((val) == (val));

#define __shfl_xor(var, lane) __shfl_xor_sync(0xffff, var, lane, 32)
#define __shfl_up(var, lane) __shfl_up_sync(0xffff, var, lane, 32)

typedef unsigned short uint16;

struct Jparticle{
	float3 pos;
	float  mass;
	float3 vel;
	float  pad;
	Jparticle() {}
	Jparticle(double mj, double xj[3], double vj[3]){
		pos.x = xj[0];
		pos.y = xj[1];
		pos.z = xj[2];
		mass  = mj;
		vel.x = vj[0];
		vel.y = vj[1];
		vel.z = vj[2];

		NAN_CHECK(xj[0]);
		NAN_CHECK(xj[1]);
		NAN_CHECK(xj[2]);
		NAN_CHECK(mj);
		NAN_CHECK(vj[0]);
		NAN_CHECK(vj[1]);
		NAN_CHECK(vj[2]);
	}
	__device__
	Jparticle(const float4 *buf){
		float4 tmp1 = buf[0];
		float4 tmp2 = buf[1];
		pos.x = tmp1.x;
		pos.y = tmp1.y;
		pos.z = tmp1.z;
		mass  = tmp1.w;
		vel.x = tmp2.x;
		vel.y = tmp2.y;
		vel.z = tmp2.z;
	}
};
struct Iparticle{
	float3 pos;
	float  h2;
	float3 vel;
	float  dtr;
	Iparticle() {}
	Iparticle(double h2i, double dtri, double xi[3], double vi[3]){
		pos.x = xi[0];
		pos.y = xi[1];
		pos.z = xi[2];
		h2    = h2i;
		vel.x = vi[0];
		vel.y = vi[1];
		vel.z = vi[2];
		dtr   = dtri;

		NAN_CHECK(xi[0]);
		NAN_CHECK(xi[1]);
		NAN_CHECK(xi[2]);
		NAN_CHECK(h2i);
		NAN_CHECK(vi[0]);
		NAN_CHECK(vi[1]);
		NAN_CHECK(vi[2]);
	}
};
struct Force{
	float3 acc;
	float  pot;
	float3 jrk;
	int    nnb;          //  8 words
	__device__  void clear(){
		acc.x = acc.y = acc.z = 0.f;
		jrk.x = jrk.y = jrk.z = 0.f;
		pot = 0.f;
		nnb = 0;
	}
	__device__ void operator+=(const Force &rhs){
		acc.x += rhs.acc.x;
		acc.y += rhs.acc.y;
		acc.z += rhs.acc.z;
		pot   += rhs.pot;
		jrk.x += rhs.jrk.x;
		jrk.y += rhs.jrk.y;
		jrk.z += rhs.jrk.z;
		if(nnb>=0 && rhs.nnb>=0){
			nnb += rhs.nnb;
		}else{
			nnb = -1;
		}
	}
#if __CUDA_ARCH__ >= 300
	__device__ void reduce_with(const int mask){
		acc.x += __shfl_xor(acc.x, mask);
		acc.y += __shfl_xor(acc.y, mask);
		acc.z += __shfl_xor(acc.z, mask);
		pot   += __shfl_xor(pot  , mask);
		jrk.x += __shfl_xor(jrk.x, mask);
		jrk.y += __shfl_xor(jrk.y, mask);
		jrk.z += __shfl_xor(jrk.z, mask);
		int ntmp = __shfl_xor(nnb, mask);
		if(nnb>=0 && ntmp>=0){
			nnb += ntmp;
		}else{
			nnb = -1;
		}
	}
#endif
};

__device__ void dev_gravity(
		const int        jidx,
		const Iparticle &ip, 
		const Jparticle &jp, 
		__out Force     &fo,
		__out uint16     nblist[]){
	float dx = jp.pos.x - ip.pos.x;
	float dy = jp.pos.y - ip.pos.y;
	float dz = jp.pos.z - ip.pos.z;
	float dvx = jp.vel.x - ip.vel.x;
	float dvy = jp.vel.y - ip.vel.y;
	float dvz = jp.vel.z - ip.vel.z;

	float r2 = dx*dx + dy*dy + dz*dz;
#if 1
	float dxp = dx + ip.dtr * dvx;
	float dyp = dy + ip.dtr * dvy;
	float dzp = dz + ip.dtr * dvz;
	float r2p = dxp*dxp + dyp*dyp + dzp*dzp;
#else
	float r2p = r2;
#endif
	float rv = dx*dvx + dy*dvy + dz*dvz;
	float rinv1 = rsqrtf(r2);
 	if(min(r2, r2p)  < jp.mass * ip.h2){
 		// fo.neib[fo.nnb++ % NBMAX] = j;
 		nblist[fo.nnb & (NNB_PER_BLOCK-1)] = (uint16)jidx;
 		fo.nnb++;
		rinv1 = 0.f;
	}
	float rinv2 = rinv1 * rinv1;
	float mrinv1 = jp.mass * rinv1;
	float mrinv3 = mrinv1 * rinv2;
	rv *= -3.f * rinv2;
	
#ifdef POTENTIAL
	fo.pot += mrinv1;
#endif
	fo.acc.x += mrinv3 * dx;
	fo.acc.y += mrinv3 * dy;
	fo.acc.z += mrinv3 * dz;
	// fo.acc.z += 1.0;
	fo.jrk.x += mrinv3 * (dvx + rv * dx);
	fo.jrk.y += mrinv3 * (dvy + rv * dy);
	fo.jrk.z += mrinv3 * (dvz + rv * dz);
}

__global__ void gravity_kernel(
		const int       nbody,
		const Iparticle ipbuf[],
		const Jparticle jpbuf[],
		__out Force     fobuf[][NJBLOCK],
		__out uint16    nbbuf[][NJBLOCK][NNB_PER_BLOCK]){
	int ibid = blockIdx.x;
	int jbid = blockIdx.y;
	int tid = threadIdx.x;
	int iaddr = tid + blockDim.x * ibid;
	int jstart = (nbody * (jbid  )) / NJBLOCK;
	int jend   = (nbody * (jbid+1)) / NJBLOCK;

	Iparticle ip = ipbuf[iaddr];
	Force fo;
	fo.clear();
	uint16 *nblist = nbbuf[iaddr][jbid];
#if __CUDA_ARCH__ >= 300 // just some trial
	for(int j=jstart; j<jend; j+=32){
		__shared__ Jparticle jpshare[32];
		__syncthreads();
		float4 *src = (float4 *)&jpbuf[j];
		float4 *dst = (float4 *)jpshare;
		dst[tid] = src[tid];
		__syncthreads();
		if(jend-j < 32){
#pragma unroll 4
			for(int jj=0; jj<jend-j; jj++){
				const Jparticle jp = jpshare[jj];
				// const Jparticle jp( (float4 *)jpshare + 2*jj);
				dev_gravity(j-jstart+jj, ip, jp, fo, nblist);
			}
		}else{
#pragma unroll 8
			for(int jj=0; jj<32; jj++){
				const Jparticle jp = jpshare[jj];
				// const Jparticle jp( (float4 *)jpshare + 2*jj);
				dev_gravity(j-jstart+jj, ip, jp, fo, nblist);
			}
		}
	}
#else
	for(int j=jstart; j<jend; j+=NTHREAD){
		__shared__ Jparticle jpshare[NTHREAD];
		__syncthreads();
		float4 *src = (float4 *)&jpbuf[j];
		float4 *dst = (float4 *)jpshare;
		dst[        tid] = src[        tid];
		dst[NTHREAD+tid] = src[NTHREAD+tid];
		__syncthreads();

		if(jend-j < NTHREAD){
#pragma unroll 4
			for(int jj=0; jj<jend-j; jj++){
				Jparticle jp = jpshare[jj];
				dev_gravity(j-jstart+jj, ip, jp, fo, nblist);
			}
		}else{
#pragma unroll 8
			for(int jj=0; jj<NTHREAD; jj++){
				Jparticle jp = jpshare[jj];
				dev_gravity(j-jstart+jj, ip, jp, fo, nblist);
			}
		}
	}
#endif
	if(fo.nnb > NNB_PER_BLOCK) fo.nnb = -1;
	fobuf[iaddr][jbid] = fo;
}

#if __CUDA_ARCH__ >= 300
__device__ void warp_reduce_int(int inp, int *out){
	inp += __shfl_xor(inp, 1);
	inp += __shfl_xor(inp, 2);
	inp += __shfl_xor(inp, 4);
	inp += __shfl_xor(inp, 8);
# if NXREDUCE>=32
	inp += __shfl_xor(inp, 16);
# endif
	*out = inp;
}
__device__ void warp_reduce_float8(float4 inp1, float4 inp2, float *out){
#  if NXREDUCE >= 64
	const int tid = threadIdx.x % 32;
#  else
	const int tid = threadIdx.x;
#  endif
	float4 tmp4L = (4&tid) ? inp2 : inp1;
	float4 tmp4R = (4&tid) ? inp1 : inp2;
	tmp4L.x += __shfl_xor(tmp4R.x, 4);
	tmp4L.y += __shfl_xor(tmp4R.y, 4);
	tmp4L.z += __shfl_xor(tmp4R.z, 4);
	tmp4L.w += __shfl_xor(tmp4R.w, 4);
	float4 tmp4;
	tmp4.x = (2&tid) ? tmp4L.z : tmp4L.x;
	tmp4.y = (2&tid) ? tmp4L.w : tmp4L.y;
	tmp4.z = (2&tid) ? tmp4L.x : tmp4L.z;
	tmp4.w = (2&tid) ? tmp4L.y : tmp4L.w;
	tmp4.x += __shfl_xor(tmp4.z, 2);
	tmp4.y += __shfl_xor(tmp4.w, 2);
	float2 tmp2;
	tmp2.x = (1&tid) ? tmp4.y : tmp4.x;
	tmp2.y = (1&tid) ? tmp4.x : tmp4.y;
	tmp2.x += __shfl_xor(tmp2.y, 1);

	tmp2.x += __shfl_xor(tmp2.x, 8);
# if NXREDUCE>=32
	tmp2.x += __shfl_xor(tmp2.x, 16);
# endif
	if(tid < 8){
		out[tid] = tmp2.x;
	}
}
#endif

__global__ void force_reduce_kernel(
		const int ni,
		const Force fpart[][NJBLOCK],
		__out Force ftot []){
	const int xid = threadIdx.x;
	const int yid = threadIdx.y;
	const int bid = blockIdx.x;
	const int iaddr = yid + blockDim.y * bid;

#if 1 && __CUDA_ARCH__ >= 300
	Force f;
	if(xid < NJBLOCK){
		f = fpart[iaddr][xid];
	}else{
		f.clear();
	}

#  if NXREDUCE >= 64
#   warning "experimental"
	__shared__ Force fshare[NYREDUCE][NXREDUCE/32];
	Force *fs = &fshare[yid][xid/32];
	if(iaddr < ni){
		const float4 tmp1 = make_float4(f.acc.x, f.acc.y, f.acc.z, f.pot);
		const float4 tmp2 = make_float4(f.jrk.x, f.jrk.y, f.jrk.z, 0.0f);
		const int    itmp = f.nnb;
		float *dst  = &(fs->acc.x);
		int   *idst = &(fs->nnb);
		warp_reduce_float8(tmp1, tmp2, dst);
		warp_reduce_int(itmp, idst);
#    if NXREDUCE==64
		__syncthreads();
		if(0 == threadIdx.x){
			Force fout = fs[0];
			fout += fs[1];
			ftot[iaddr] = fout;
		}
#    elif NXREDUCE==128
		__syncthreads();
		if(0 == threadIdx.x){
			Force f01 = fs[0];
			f01 += fs[1];

			Force f23 = fs[2];
			f23 += fs[3];

			f01 += f23;

			ftot[iaddr] = f01;
		}
#    else
#      error
#    endif
	}
#  else // 32 thread version
	if(iaddr < ni){
		const float4 tmp1 = make_float4(f.acc.x, f.acc.y, f.acc.z, f.pot);
		const float4 tmp2 = make_float4(f.jrk.x, f.jrk.y, f.jrk.z, 0.0f);
		const int    itmp = f.nnb;
		float *dst  = (float *)(ftot + iaddr);
		int   *idst = (int *)(dst + 7);
		warp_reduce_float8(tmp1, tmp2, dst);
		warp_reduce_int(itmp, idst);
	}
#  endif
#else // usual shared memory version
	__shared__ Force fshare[NYREDUCE][NXREDUCE];
	if(xid < NJBLOCK){
		fshare[yid][xid] = fpart[iaddr][xid];
	}else{
		fshare[yid][xid].clear();
	}
	Force *fs = fshare[yid];

#  if NXREDUCE>=64
	__syncthreads();
#  endif
#  if NXREDUCE>=128
	if(xid < 64) fs[xid] += fs[xid + 64];
	__syncthreads();
#  endif
#  if NXREDUCE>=64
	if(xid < 32) fs[xid] += fs[xid + 32];
	__syncthreads();
#  endif
#  if NXREDUCE>=32
	if(xid < 16) fs[xid] += fs[xid + 16];
#  endif
	if(xid < 8) fs[xid] += fs[xid + 8];
	if(xid < 4) fs[xid] += fs[xid + 4];
	if(xid < 2) fs[xid] += fs[xid + 2];
	if(xid < 1) fs[xid] += fs[xid + 1];
	
	// if(iaddr < ni){
	if(iaddr < ni && xid==0){  // for NXREDUCE > 32
		ftot[iaddr] = fs[0];
	}
#endif
}

__global__ void gather_nb_kernel(
		const int    ni,
		const int    nj,
		const int    joff,
		const Force  fpart[][NJBLOCK],
		const Force  ftot [],
		const int    nboff[],
		const uint16 nbpart[][NJBLOCK][NNB_PER_BLOCK],
		__out   int  nblist[])
{
	const int xid = threadIdx.x;
	const int yid = threadIdx.y;
	const int bid = blockIdx.x;
	const int iaddr = yid + blockDim.y * bid;
	if(iaddr >= ni) return;
	if(ftot[iaddr].nnb < 0) return;

	const int mynnb = (xid < NJBLOCK) ? fpart[iaddr][xid].nnb
	                                  : 0;

	// now performe prefix sum
#if 1 ||  __CUDA_ARCH__ >= 300
	int ix = mynnb;
#if NXREDUCE<=32
	#pragma unroll
	for(int ioff=1; ioff<NXREDUCE; ioff*=2){
		int iy = __shfl_up(ix, ioff);
		if(xid>=ioff) ix += iy;
	}
	int iz = __shfl_up(ix, 1);
	const int off = (xid == 0) ? 0 : iz;
#else
	#pragma unroll
	for(int ioff=1; ioff<32; ioff*=2){
		int iy = __shfl_up(ix, ioff);
		if(xid%32>=ioff) ix += iy;
	}
	__shared__ int ishare[NYREDUCE][NXREDUCE];
	volatile int *ish = ishare[yid];
	ish[xid] = ix;
	__syncthreads();
#    if NXREDUCE==64
	if(xid >= 32){
		ish[xid] += ish[31];
	}
	__syncthreads();
#    elif NXREDUCE==128
	if(xid%64 >= 32){
		ish[xid] += ish[(xid/32*32)-1];
	}
	__syncthreads();
	if(xid >= 64){
		ish[xid] += ish[63];
	}
	__syncthreads();
#    else
#        error
#    endif
	const int off = (xid == 0) ? 0 : ish[xid-1];
#endif
#else
	__shared__ int ishare[NYREDUCE][NXREDUCE];
	ishare[yid][xid] = mynnb;
	volatile int *ish = ishare[yid];
	if(xid>=1)  ish[xid] += ish[xid-1];
	if(xid>=2)  ish[xid] += ish[xid-2];
	if(xid>=4)  ish[xid] += ish[xid-4];
	if(xid>=8)  ish[xid] += ish[xid-8];
#if NXREDUCE>=32
	if(xid>=16)  ish[xid] += ish[xid-16];
#endif
	const int off = (xid == 0) ? 0 
	                           : ish[xid-1];
#endif
	int *nbdst = nblist + nboff[iaddr] + off;

	const int jstart = (nj * xid) / NJBLOCK;
	if(xid < NJBLOCK){
		for(int k=0; k<mynnb; k++){
			const int nbid = (joff + jstart) + int(nbpart[iaddr][xid][k]);
#if 1
			nbdst[k] = nbid;
#else
			nbdst[k] = nbid + 1000000*xid;
#endif
		}
	}
}


// Host Part
#ifdef PROFILE
#include <sys/time.h>
static double get_wtime(){
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec + 1.e-6 * tv.tv_usec;
}
#else
static double get_wtime(){
	return 0.0;
}
#endif

static double time_send, time_grav, time_reduce;
static long long numInter;
static long ncall_send = 0;
static cudaPointer <Jparticle> jpbuf[MAX_GPU];
static cudaPointer <Iparticle> ipbuf[MAX_GPU];
static cudaPointer <Force[NJBLOCK]> fpart[MAX_GPU];
static cudaPointer <Force>          ftot [MAX_GPU];
static cudaPointer <uint16[NJBLOCK][NNB_PER_BLOCK]> nbpart[MAX_GPU];
static cudaPointer <int> nblist [MAX_GPU];
static cudaPointer <int> nboff  [MAX_GPU];
static int numCPU, numGPU;
static int joff[MAX_GPU + 1];
static int nbody, nbodymax;
static int devid[MAX_GPU];
static bool is_open = false;
static bool devinit = false;

void GPUNB_devinit(){
	if(devinit) return;

	assert(NXREDUCE >= NJBLOCK);
	// assert(NXREDUCE <= 32);
	assert(NXREDUCE <= 128);

	hipGetDeviceCount(&numGPU);
	assert(numGPU <= MAX_GPU);
	char *gpu_list = getenv("GPU_LIST");
	if(gpu_list){
		// get GPU list from environment variable
		numGPU = 0;
		char *p = strtok(gpu_list, " ");
		while(p){
			devid[numGPU++] = atoi(p);
			p = strtok(NULL, " ");
			assert(numGPU <= MAX_GPU);
		}
	}else{
		// use all GPUs
		for(int i=0; i<numGPU; i++){
			devid[i] = i;
		}
	}
	
	// numGPU = 1;
#pragma omp parallel
	{
		int tid = omp_get_thread_num();
		if(tid == 0) numCPU = omp_get_num_threads();
	}
	assert(numCPU <= MAX_CPU);
	assert(numGPU <= numCPU);
#pragma omp parallel
	{
		int tid = omp_get_thread_num();
		if(tid < numGPU){
			hipSetDevice(devid[tid]);
		}
	}
#ifdef PROFILE
	fprintf(stderr, "***********************\n");
	fprintf(stderr, "Initializing NBODY6/GPU library\n");
	fprintf(stderr, "#CPU %d, #GPU %d\n", numCPU, numGPU);
	fprintf(stderr, " device:");
	for(int i=0; i<numGPU; i++){
		fprintf(stderr, " %d", devid[i]);
	}
	fprintf(stderr, "\n");
#if 1
	for(int i=0; i<numGPU; i++){
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, devid[i]);
		fprintf(stderr, " device %d: %s\n", devid[i], prop.name);
	}
#endif
	fprintf(stderr, "***********************\n");
#endif
	devinit = true;
}

void GPUNB_open(int nbmax){
	time_send = time_grav = time_reduce = 0.0;
	numInter = 0;
	ncall_send = 0;
	nbodymax = nbmax;

	GPUNB_devinit();

	if(is_open){
		fprintf(stderr, "gpunb: it is already open\n");
		return;
	}
	is_open = true;


	for(int id=0; id<numGPU + 1; id++){
		joff[id] = (id * nbmax) / numGPU;
	}

	// omp_set_num_threads(numGPU);
#pragma omp parallel
	{
		int tid = omp_get_thread_num();
		if(tid < numGPU){
			hipSetDevice(devid[tid]);
			int nj = joff[tid+1] - joff[tid];
			jpbuf [tid].allocate(nj + NTHREAD);
			ipbuf [tid].allocate(NIMAX);
			fpart [tid].allocate(NIMAX);
			ftot  [tid].allocate(NIMAX);
			nbpart[tid].allocate(NIMAX);
			nblist[tid].allocate(NB_BUF_SIZE); // total ganged nblist
			nboff [tid].allocate(NIMAX+1);
		}
	}
#ifdef PROFILE
	fprintf(stderr, "***********************\n");
	fprintf(stderr, "Opened NBODY6/GPU library\n");
	fprintf(stderr, "#CPU %d, #GPU %d\n", numCPU, numGPU);
	fprintf(stderr, " device:");
	for(int i=0; i<numGPU; i++){
		fprintf(stderr, " %d", devid[i]);
	}
	fprintf(stderr, "\n");
	for(int i=0; i<numGPU+1; i++){
		fprintf(stderr, " %d", joff[i]);
	}
	fprintf(stderr, "\n");
	fprintf(stderr, "nbmax = %d\n", nbmax);
	fprintf(stderr, "***********************\n");
#endif
}

void GPUNB_close(){
	if(!is_open){
		fprintf(stderr, "gpunb: it is already close\n");
		return;
	}
	is_open = false;
	// omp_set_num_threads(numGPU);
#pragma omp parallel
	{
		int tid = omp_get_thread_num();
		if(tid < numGPU){
			jpbuf [tid].free();
			ipbuf [tid].free();
			fpart [tid].free();
			ftot  [tid].free();
			nbpart[tid].free();
			nblist[tid].free();
			nboff [tid].free();
		}
	}
	// omp_set_num_threads(numCPU);
	nbodymax = 0;

#ifdef PROFILE
	double byte_sent = (double)ncall_send * nbody * sizeof(Jparticle);
	double GB_per_sec = byte_sent / time_send * 1.e-9;
	fprintf(stderr, "***********************\n");
	fprintf(stderr, "Closed NBODY6/GPU library\n");
	fprintf(stderr, "time send   : %f sec\n", time_send);
	fprintf(stderr, "time grav   : %f sec\n", time_grav);
	fprintf(stderr, "time reduce : %f sec\n", time_reduce);
	fprintf(stderr, "time regtot : %f sec\n", time_send + time_grav + time_reduce);
	fprintf(stderr, "%f Gflops (gravity part only)\n", 60.e-9 * numInter / time_grav);
	fprintf(stderr, "%f GB/s for send j-particle\n", GB_per_sec);
	fprintf(stderr, "***********************\n");
#endif
}

void GPUNB_send(
		int _nbody,
		double mj[],
		double xj[][3],
		double vj[][3]){
	assert(is_open);
	nbody = _nbody;
	assert(nbody <= nbodymax);
	time_send -= get_wtime();
	for(int id=0; id<numGPU + 1; id++){
		joff[id] = (id * nbody) / numGPU;
	}
#pragma omp parallel
	{
		int tid = omp_get_thread_num();
#if 0
		if(tid < numGPU){
			int nj = joff[tid+1] - joff[tid];
			for(int j=0; j<nj; j++){
				int jj = j + joff[tid];
				jpbuf[tid][j] = Jparticle(mj[jj], xj[jj], vj[jj]);
			}
			jpbuf[tid].htod(nj);
		}
#else // use all CPU cores for data packing
		for(int ig=0; ig<numGPU; ig++){
			int nj = joff[ig+1] - joff[ig];
			#pragma omp for nowait
			for(int j=0; j<nj; j++){
				int jj = j + joff[ig];
				jpbuf[ig][j] = Jparticle(mj[jj], xj[jj], vj[jj]);
			}
		}
		#pragma omp barrier
		if(tid < numGPU){
			int nj = joff[tid+1] - joff[tid];
			jpbuf[tid].htod(nj);
		}
#endif
	}
	time_send += get_wtime();
	ncall_send++;
}

void GPUNB_regf(
		int ni,
		double h2[],
		double dtr[],
		double xi[][3],
		double vi[][3],
		double acc[][3],
		double jrk[][3],
		double pot[],
		int lmax,
		int nnbmax,
		int *listbase){
	assert(is_open);

	time_grav -= get_wtime();
	numInter += ni * nbody;
	assert(0 < ni && ni <= NIMAX);

	// omp_set_num_threads(numGPU);
#pragma omp parallel
	{
		int tid = omp_get_thread_num();
		if(tid < numGPU){
			// hipSetDevice(device_id[tid]);
			for(int i=0; i<ni; i++){
				ipbuf[tid][i] = Iparticle(h2[i], dtr[i], xi[i], vi[i]);
			}
			// set i-particles
			ipbuf[tid].htod(ni);

			// gravity kernel
			int niblock = 1 + (ni-1) / NTHREAD;
			dim3 grid(niblock, NJBLOCK, 1);
			dim3 threads(NTHREAD, 1, 1);
			int nj = joff[tid+1] - joff[tid];
			gravity_kernel <<< grid, threads >>> 
				(nj, ipbuf[tid], jpbuf[tid], fpart[tid], nbpart[tid]);
			// CUDA_SAFE_THREAD_SYNC();

#if 0
			dim3 rgrid(niblock, 1, 1);
			reduce_kernel <<< rgrid, threads >>>
				(nj, joff[tid], fpart[tid], nbpart[tid], ftot[tid], nbtot[tid]);
#else
			const int ni8 = 1 + (ni-1) / NYREDUCE;
			dim3 rgrid   (ni8, 1, 1);
			dim3 rthreads(NXREDUCE, NYREDUCE, 1);
			force_reduce_kernel <<< rgrid, rthreads >>>
				(ni, fpart[tid], ftot[tid]);
#endif
			// CUDA_SAFE_THREAD_SYNC();
			ftot [tid].dtoh(ni);

#if 0
			// DEBUG
			for(int i=0; i<ni; i++){
				printf("%d %d %10.4e\n", i, ftot[0][i].nnb, ftot[0][i].acc.x);
			}
			fflush(stdout);
			exit(1);
#endif

			// now make prefix sum
			int nbsum = 0;
			for(int i=0; i<ni; i++){
				nboff[tid][i] = nbsum;
				const int nnb = ftot[tid][i].nnb;
				// assert(nnb >= 0);
				if(nnb >= 0) nbsum += nnb;
			}
			assert(nbsum <= NB_BUF_SIZE);
			nboff[tid].htod(ni);

			// debugging
			// for(int k=0; k<nbsum; k++) nblist[tid][k] = -1;
			// nblist[tid].htod(nbsum);

			gather_nb_kernel <<< rgrid, rthreads>>>
				(ni, nj, joff[tid], fpart[tid], ftot[tid], 
				 nboff[tid], nbpart[tid], nblist[tid]);
			// CUDA_SAFE_THREAD_SYNC();
			nblist[tid].dtoh(nbsum);
#if 0
			// DEBUG
			for(int i=0; i<64; i++){
				const int nnb = ftot[0][i].nnb;
				int off = 0;

				printf("%d : %d :", i, nnb);
				for(int j=0; j<nnb; j++){
					// printf(" %d:%d", j, nblist[0][j + off]);
					printf(" %d", nblist[0][j + off]);
				}
				printf("\n");

				off += nnb;
			}
			fflush(stdout);
			exit(1);
#endif
		}
	}

	const double wt = get_wtime();
	time_grav   += wt;
	time_reduce -= wt;

	// reduction phase
	// omp_set_num_threads(numCPU);
#pragma omp parallel for
	for(int i=0; i<ni; i++){
		double ax=0.0, ay=0.0, az=0.0;
		double jx=0.0, jy=0.0, jz=0.0;
		double po=0.0;

		for(int id=0; id<numGPU; id++){
			Force &fo = ftot[id][i];
			ax += fo.acc.x;
			ay += fo.acc.y;
			az += fo.acc.z;
			jx += fo.jrk.x;
			jy += fo.jrk.y;
			jz += fo.jrk.z;
			po += fo.pot;
		}
		acc[i][0] = ax;
		acc[i][1] = ay;
		acc[i][2] = az;
		jrk[i][0] = jx;
		jrk[i][1] = jy;
		jrk[i][2] = jz;
		pot[i]    = po;
	}
#pragma omp parallel for
	for(int i=0; i<ni; i++){
		bool overflow = false;
		int *nnbp = listbase + lmax * i;
		int *nblistp = nnbp + 1;
		int nnb = 0;
		for(int id=0; id<numGPU; id++){
			const int nnb_part = ftot[id][i].nnb;
			if(nnb_part < 0){
				overflow = true;
				fprintf(stderr, "!!!overflow : i=%d, id=%d, nnb_part=%d\n", i, id, nnb_part);
			}
			// assert(!overflow);
			nnb += nnb_part;
			if(nnb > nnbmax){
				overflow = true;
				fprintf(stderr, "!!!overflow : i=%d, id=%d, nnb_tot =%d, nnbmax=%d\n", i, id, nnb, nnbmax);
			}
			// assert(!overflow);
			if(!overflow){
				const int off = nboff[id][i]; 
				for(int k=0; k<nnb_part; k++){
					*nblistp++ = nblist[id][off + k];
				}
			}
		}
		if(overflow){
			// *nnbp = -1;
			*nnbp = nnb ? -abs(nnb) : -9999;
		}else{
			*nnbp = nnb;
		}
	}
	time_reduce += get_wtime();
}

extern "C" {
	void gpunb_devinit_(){
		GPUNB_devinit();
	}
	void gpunb_open_(int *nbmax){
		GPUNB_open(*nbmax);
	}
	void gpunb_close_(){
		GPUNB_close();
	}
	void gpunb_send_(
			int *nj,
			double mj[],
			double xj[][3],
			double vj[][3]){
		GPUNB_send(*nj, mj, xj, vj);
	}
	void gpunb_regf_(
			int *ni,
			double h2[],
			double dtr[],
			double xi[][3],
			double vi[][3],
			double acc[][3],
			double jrk[][3],
			double pot[],
			int *lmax,
			int *nbmax,
			int *list){ // list[][lmax]
		GPUNB_regf(*ni, h2, dtr, xi, vi, acc, jrk, pot, *lmax, *nbmax, list);
	}
}

